﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "C:\Users\crazy\source\repos\AI\AI\Matrix.cuh"

int main()
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Create two matrices
    Matrix A(2, 2);
    Matrix B(2, 2);

    // Fill the matrices with some values
    A(0, 0) = 1.0f; A(0, 1) = 2.0f;
    A(1, 0) = 3.0f; A(1, 1) = 4.0f;

    B(0, 0) = 5.0f; B(0, 1) = 6.0f;
    B(1, 0) = 7.0f; B(1, 1) = 8.0f;

    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        // Handle error...
    }



    // Multiply the matrices
    Matrix C = A + B;

     err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        // Handle error...
    }



    A.print();
    B.print();

    // Wait for GPU to finish before exiting
    hipDeviceSynchronize();

    return 0;
}



